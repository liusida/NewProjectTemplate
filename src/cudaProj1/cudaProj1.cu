#include "hip/hip_runtime.h"
#include "stdio.h"
#include "cudaProj1.h"

__global__
void Kernel() {
    int i = threadIdx.x;
    if (i==2) {
        printf("Testing debugging!\n");
    }
    printf("in Kernel %d.\n", i);
}

cudaProj1::cudaProj1(/* args */)
{
}

cudaProj1::~cudaProj1()
{
}

void cudaProj1::startKernel() {
    Kernel<<<1,3>>>();
    hipDeviceSynchronize();
}